#include <iostream>

int main(void) {
  hipDeviceProp_t prop;
  
  int count;
  hipGetDeviceCount(&count);
  
  printf("%d Cuda devices found!\n", count);
  
  for (int i=0; i<count; i++) {
    hipGetDeviceProperties(&prop, i);
    printf( "--- General Information for device %d ---\n", i );
    printf( "Name:%s\n", prop.name );
    printf("Compute capability: %d. %d\n", prop.major, prop.minor);
    printf("Clock rate: %d\n", prop.clockRate);
    printf("Device copy overlap:  ");
    if (prop.deviceOverlap)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
      
    printf("Kernel execution timeout: ");
    if (prop.kernelExecTimeoutEnabled)
      printf( "Enabled\n" );
    else
      printf( "Disabled\n" );
      
    printf("Total global mem: %ld\n", prop.totalGlobalMem);
    printf("Total constant mem: %ld\n", prop.totalConstMem);
    printf("Max mem pitch: %ld\n", prop.memPitch);
      
      
      
    printf( "Multiprocessor count:  %d\n", prop.multiProcessorCount );
    printf("Registers per mp:  %d\n", prop.regsPerBlock);
    printf("Threads in warp: %d\n", prop.warpSize);
    printf( "Max threads per block: %d\n", prop.maxThreadsPerBlock );
    printf( "Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2] );
    printf( "Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2] );
  }
}