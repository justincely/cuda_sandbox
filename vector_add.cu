
#include <hip/hip_runtime.h>
#include <iostream>    // Needed to perform IO operations
using namespace std;

#define N 100000

__global__ void add(int n, int *a, int *b, int *c) {
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
  
  for (int i = index; i < n; i += stride) {
    c[i] = a[i] + b[i];
  }
  
}

int main(void) {
  int blockSize = 256;
  int numBlocks = (N + blockSize -1) / blockSize;
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;
  
  hipMalloc((void**)&dev_a, N*sizeof(int));
  hipMalloc((void**)&dev_b, N*sizeof(int));
  hipMalloc((void**)&dev_c, N*sizeof(int));

  for (int i=0; i<N; i++) {
    a[i] = -i;
    b[i] = i*i;
  }
  
  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
  
  add<<<numBlocks, blockSize>>>(N, dev_a, dev_b, dev_c);
  
  hipDeviceSynchronize();
  
  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
  
  for (int i=0; i<N; i++) {
    printf( "%d + %d = %d\n", a[i], b[i], c[i] );
  }
  
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  
  return 0;
}